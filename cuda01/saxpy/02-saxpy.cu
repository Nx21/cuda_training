#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 200,000 ns.
 *
 * Some bugs have been placed in this codebase for your edification.
 */
__global__
void initWith(int num, int *a)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__ void saxpy(int * a, int * b, int * c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < N; i += stride)
        c[i] = 2 * a[i] + b[i];
}

int main()
{
    int deviceId;
    int numberOfSMs;
    hipError_t addArraysErr;
    hipError_t asyncErr;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);
    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);
    int threadsPerBlock = 128;
    int numberOfBlocks = 32 * numberOfSMs;
    initWith<<<numberOfBlocks, threadsPerBlock>>>(2, a);
    initWith<<<numberOfBlocks, threadsPerBlock>>>(1, b);
    initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c);

    saxpy <<< numberOfBlocks, threadsPerBlock >>> ( a, b, c );
    addArraysErr = hipGetLastError();
    if(addArraysErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addArraysErr));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
  
    hipMemPrefetchAsync(c, size, hipCpuDeviceId);
    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
   

    hipFree( a ); hipFree( b ); hipFree( c );
}
